#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/logical.h>

#include <vector>
#include <iostream>

#include "util.h"

struct Row {
    int rel;

    int l, r;

    int from, to;

    bool learning;

    __host__ __device__
    Row() : rel(0), l(0), r(0), from(0), to(0), learning(true) {}

    __device__
    Row(int rel, int cos, int size) {
        l = 0;
        r = size - 1;
        
        from = to = cos;
        
        this->rel = rel;

        learning = true;
    }
};

std::ostream &operator<<(std::ostream &o, const Row &r) {
    return o << "Row[" << r.rel << "]{" << r.l << ":" << r.from << "-" << r.to << ":" << r.r << "}(" << r.learning << ")";
}

struct Rel {
    int gens[2];
    int mul;
};

struct Solver {
    int ngens;
    int *cosets;
    Rel *rels;
    
    Solver(int ngens,
           thrust::device_vector<int> &cosets,
           thrust::device_vector<Rel> &rels)
        : ngens(ngens),
           cosets(thrust::raw_pointer_cast(cosets.data())),
           rels(thrust::raw_pointer_cast(rels.data())) {
        }
    
    __device__
    void operator()(Row &r) {
        if (r.r - r.l <= 1) {
            r.learning = false;
            return;
        }
        
        while (r.r - r.l > 1) {
            int gen = rels[r.rel].gens[r.l & 1];
            int next = cosets[r.from * ngens + gen];
            if (next < 0) break;
            r.l++;
            r.from = next;
        }

        while (r.r - r.l > 1) {
            int gen = rels[r.rel].gens[r.r & 1];
            int next = cosets[r.to * ngens + gen];
            if (next < 0) break;
            r.r--;
            r.to = next;
        }
            
        if (r.r - r.l <= 1) { 
            int gen = rels[r.rel].gens[r.l & 1];
            cosets[r.from * ngens + gen] = r.to;
            cosets[r.to * ngens + gen] = r.from;

            r.learning = true;
            return;
        }

        r.learning = false;
    }
};

struct CosetInitializer {
    int *cosets;

    CosetInitializer(thrust::device_vector<int> &cosets)
        : cosets(thrust::raw_pointer_cast(cosets.data())) {
    }

    __device__
    void operator()(int gen) {
        cosets[gen] = 0;
    }
};

struct RowGen {
    Rel *rels;

    int coset;

    RowGen(int coset, thrust::device_vector<Rel> &rels) 
        : coset(coset),
          rels(thrust::raw_pointer_cast(rels.data())) {}

    __device__
    Row operator()(int rel) {
        return Row(rel, coset, rels[rel].mul * 2);
    }
};

struct RowIncomplete {
    __device__
    bool operator()(Row r) {
        return r.r - r.l > 1;
    }
};

struct Relearn {
    __device__
    void operator()(Row &r) {
        r.learning = true;
    }
};

struct Learning {
    __device__
    bool operator()(Row r) {
        return r.learning;
    }
};

void add_row(
        int ngens,
        thrust::device_vector<int> &cosets) {
    cosets.resize(cosets.size() + ngens, -1);
};

// todo: this part is _real_ slow.
bool add_coset(
        int ngens,
        int *coset,
        int *hint,
        thrust::device_vector<int> &cosets) {
    *coset = cosets.size() / ngens;

    // todo: this part especially.
    while (cosets[*hint] >= 0) {
        *hint = *hint + 1;
        if (*hint >= cosets.size()) 
            return true;
    }
    int from = *hint / ngens;
    int gen = *hint % ngens;
    
    add_row(ngens, cosets);
    
    cosets[*hint] = *coset;
    cosets[*coset * ngens + gen] = from;

    return false;
}

void gen_rows(
        int coset,
        thrust::device_vector<Rel> &rels,
        thrust::device_vector<Row> &rows) {
    rows.resize(rows.size() + rels.size());

    thrust::counting_iterator<int> counter(0);
    thrust::transform(
            thrust::device,
            counter, counter + rels.size(),
            rows.end() - rels.size(),
            RowGen(coset, rels));
}

thrust::device_vector<int> solve(
        int ngens,
        thrust::device_vector<int> subs,
        thrust::device_vector<Rel> rels) {
    
    thrust::device_vector<int> cosets;
    thrust::device_vector<Row> rows;

    add_row(ngens, cosets);
    thrust::for_each(
            thrust::device,
            subs.begin(), subs.end(), 
            CosetInitializer(cosets));

    gen_rows(0, rels, rows);

    int coset = 0;
    int hint = 0;

    while (true) {
        thrust::for_each(
                thrust::device, 
                rows.begin(), 
                rows.end(),
                Relearn());

        Solver solve(ngens, cosets, rels);
        while (true) {
            thrust::for_each(
                    thrust::device,
                    rows.begin(), rows.end(),
                    solve);
            bool r = thrust::any_of(
                    thrust::device,
                    rows.begin(), rows.end(),
                    Learning());
            if (!r) break;
        }

        bool done = add_coset(
                ngens,
                &coset, &hint,
                cosets);

        if (done) break;

        gen_rows(coset, rels, rows);

        auto cut = thrust::partition(
                thrust::device, 
                rows.begin(), rows.end(), 
                RowIncomplete());

        rows.erase(cut, rows.end());
    }

    return cosets;
}


int main(int argc, char* argv[]) {
    // int ngens = 4;
    // std::vector<Rel> rels = {
    //     {0, 1, 3},
    //     {1, 2, 3},
    //     {2, 3, 3},

    //     {0, 2, 2},
    //     {1, 2, 2},
    //     {1, 3, 2},
    // };
    // std::vector<int> subs = {};

    int ngens = 4;
    std::vector<Rel> rels = {
        {0, 1, 4},
        {1, 2, 3},
        {2, 3, 3},

        {0, 2, 2},
        {1, 3, 2},
        {0, 3, 2},
    };
    std::vector<int> subs = {};

    thrust::host_vector<int> cosets = solve(ngens, subs, rels);

    std::cout << cosets.size() / ngens << " cosets" << std::endl;
    for (int c = 0; c < cosets.size(); c += ngens) {
        for (int g = c; g < c + ngens; g++ ) {
            std::cout << cosets[g] << " ";
        } std::cout << std::endl;
    }

    return 0;
}

