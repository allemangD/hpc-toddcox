#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>

#include <vector>
#include <iostream>

#include "util.h"

// struct Cosets {
//     int width;
//     thrust::device_vector<int> data{};
//     
//     __host__
//     Cosets(int ngens) : width(ngens) {
//     }
//     
//     void add_row() {
//         data.resize(data.size() + width, -1);
//     }
//     
//     thrust::host_vector<int> get_data() {
//         return data;
//     }
// };

struct Row {
    int rel;

    int l, r;

    int from, to;

    __host__ __device__
    Row() : rel(0), l(0), r(0), from(0), to(0) {}

    __device__
    Row(int rel, int cos, int size) {
        l = 0;
        r = size - 1;
        
        from = to = cos;
        
        this->rel = rel;
    }
};

std::ostream &operator<<(std::ostream &o, const Row &r) {
    return o << "Row[" << r.rel << "]{" << r.l << ":" << r.from << "-" << r.to << ":" << r.r << "}";
}

struct Rel {
    int gens[2];
    int mul;
};

struct Solver {
    int ngens;
    int *cosets;
    Rel *rels;
    
    Solver(int ngens,
           thrust::device_vector<int> &cosets,
           thrust::device_vector<Rel> &rels)
        : ngens(ngens),
           cosets(thrust::raw_pointer_cast(cosets.data())),
           rels(thrust::raw_pointer_cast(rels.data())) {
        }
    
    __device__
    void operator()(Row &r) {
        if (r.l + 1 >= r.r) return;
        
        while ((r.r - r.l) > 0) {
            int gen = rels[r.rel].gens[r.l & 1];
            int next = cosets[r.from * ngens + gen];
            if (next < 0) break;
            r.l++;
            r.from = next;
        }

        while ((r.r - r.l) > 0) {
            int gen = rels[r.rel].gens[r.r & 1];
            int next = cosets[r.to * ngens + gen];
            if (next < 0) break;
            r.r--;
            r.to = next;
        }
            
        if (r.r - r.l == 0) {
            int gen = rels[r.rel].gens[r.l & 1];
            cosets[r.from * ngens + gen] = r.to;
            cosets[r.to * ngens + gen] = r.from;
        }
    }
};

struct CosetInitializer {
    int *cosets;

    CosetInitializer(thrust::device_vector<int> &cosets)
        : cosets(thrust::raw_pointer_cast(cosets.data())) {
    }

    __device__
    void operator()(int gen) {
        cosets[gen] = 0;
    }
};

struct RowGen {
    Rel *rels;

    int coset;

    RowGen(int coset, thrust::device_vector<Rel> &rels) 
        : coset(coset),
          rels(thrust::raw_pointer_cast(rels.data())) {}

    __device__
    Row operator()(int rel) {
        return Row(rel, coset, rels[rel].mul * 2);
    }
};

thrust::device_vector<int> solve(
        int ngens,
        thrust::device_vector<int> subs,
        thrust::device_vector<Rel> rels) {
    
    thrust::device_vector<int> cosets;
    cosets.resize(cosets.size() + ngens, -1);
    int lastCoset = 0;

    thrust::for_each(subs.begin(), subs.end(), 
        CosetInitializer(cosets));

    thrust::device_vector<Row> rows;

    thrust::counting_iterator<int> counter(0);

    thrust::device_vector<Row> new_rows(rels.size());
    thrust::transform(counter, counter + rels.size(), new_rows.begin(),
        RowGen(lastCoset, rels));
    rows.insert(rows.begin(), new_rows.begin(), new_rows.end());

    std::cout << rows << std::endl;

    Solver solv(ngens, cosets, rels);

    std::cout << thrust::host_vector<Row>(rows) << std::endl;
    thrust::for_each(rows.begin(), rows.end(), solv);
    std::cout << thrust::host_vector<Row>(rows) << std::endl;

    return cosets;
}


int main(int argc, char* argv[]) {
    int ngens = 4;
    std::vector<Rel> rels = {
        {0, 1, 4},
        {1, 2, 3},
        {2, 3, 3},

        {0, 2, 2},
        {1, 2, 2},
        {1, 3, 2},
    };
    std::vector<int> subs = {1, 3};

    thrust::host_vector<int> cosets = solve(ngens, subs, rels);

    std::cout << cosets << std::endl;

    return 0;
}

