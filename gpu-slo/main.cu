#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/logical.h>

#include <vector>
#include <iostream>

#include "util.h"

struct Row {
    int rel;

    int l, r;

    int from, to;

    bool learning;

    __host__ __device__
    Row() : rel(0), l(0), r(0), from(0), to(0), learning(true) {}

    __device__
    Row(int rel, int cos, int size) {
        l = 0;
        r = size - 1;
        
        from = to = cos;
        
        this->rel = rel;

        learning = true;
    }
};

std::ostream &operator<<(std::ostream &o, const Row &r) {
    return o << "Row[" << r.rel << "]{" << r.l << ":" << r.from << "-" << r.to << ":" << r.r << "}(" << r.learning << ")";
}

struct Rel {
    int gens[2];
    int mul;
};

// this performs a pass on one relation table row, applying learned data to the coset table.
struct Solver {
    int ngens;
    int *cosets;
    Rel *rels;
    
    Solver(int ngens,
           thrust::device_vector<int> &cosets,
           thrust::device_vector<Rel> &rels)
        : ngens(ngens),
           cosets(thrust::raw_pointer_cast(cosets.data())),
           rels(thrust::raw_pointer_cast(rels.data())) {
        }
    
    __device__
    void operator()(Row &r) {
        if (r.r - r.l <= 1) {
            r.learning = false;
            return;
        }
        
        while (r.r - r.l > 1) {
            int gen = rels[r.rel].gens[r.l & 1];
            int next = cosets[r.from * ngens + gen];
            if (next < 0) break;
            r.l++;
            r.from = next;
        }

        while (r.r - r.l > 1) {
            int gen = rels[r.rel].gens[r.r & 1];
            int next = cosets[r.to * ngens + gen];
            if (next < 0) break;
            r.r--;
            r.to = next;
        }
            
        if (r.r - r.l <= 1) { 
            int gen = rels[r.rel].gens[r.l & 1];
            cosets[r.from * ngens + gen] = r.to;
            cosets[r.to * ngens + gen] = r.from;

            r.learning = true;
            return;
        }

        r.learning = false;
    }
};

// this sets the inital row in the coset table based on the subgroup generators
struct CosetInitializer {
    int *cosets;

    CosetInitializer(thrust::device_vector<int> &cosets)
        : cosets(thrust::raw_pointer_cast(cosets.data())) {
    }

    __device__
    void operator()(int gen) {
        cosets[gen] = 0;
    }
};

// this creates rows for cosets by index of each relation table
struct RowGen {
    Rel *rels;

    int coset;

    RowGen(int coset, thrust::device_vector<Rel> &rels) 
        : coset(coset),
          rels(thrust::raw_pointer_cast(rels.data())) {}

    __device__
    Row operator()(int rel) {
        return Row(rel, coset, rels[rel].mul * 2);
    }
};

// determines if rows are incomplete; used to remove completed rows
struct RowIncomplete {
    __device__
    bool operator()(Row r) {
        return r.r - r.l > 1;
    }
};

// re-set rows to be learning for a next pass
struct Relearn {
    __device__
    void operator()(Row &r) {
        r.learning = true;
    }
};

// determine if rows are learning. used for exit condition
struct Learning {
    __device__
    bool operator()(Row r) {
        return r.learning;
    }
};

// add a row to the coset table filled with -1
void add_row(
        int ngens,
        thrust::device_vector<int> &cosets) {
    cosets.resize(cosets.size() + ngens, -1);
};

// add a new coset to the coset table, picking up where the last call left off.
// todo: this part is _real_ slow.
bool add_coset(
        int ngens,
        int *coset,
        int *hint,
        thrust::device_vector<int> &cosets) {
    *coset = cosets.size() / ngens;

    // todo: this part especially.
    while (cosets[*hint] >= 0) {
        *hint = *hint + 1;
        if (*hint >= cosets.size()) 
            return true;
    }
    int from = *hint / ngens;
    int gen = *hint % ngens;
    
    add_row(ngens, cosets);
    
    cosets[*hint] = *coset;
    cosets[*coset * ngens + gen] = from;

    return false;
}

// add a row for each relation table for some coset
void gen_rows(
        int coset,
        thrust::device_vector<Rel> &rels,
        thrust::device_vector<Row> &rows) {
    rows.resize(rows.size() + rels.size());

    thrust::counting_iterator<int> counter(0);
    thrust::transform(
            thrust::device,
            counter, counter + rels.size(),
            rows.end() - rels.size(),
            RowGen(coset, rels));
}

// do everything. data is implicitly passed to the device via device_vector.
thrust::device_vector<int> solve(
        int ngens,
        thrust::device_vector<int> subs,
        thrust::device_vector<Rel> rels) {
    
    thrust::device_vector<int> cosets;
    thrust::device_vector<Row> rows;

    // create the inital row and populate it from subs
    add_row(ngens, cosets);
    thrust::for_each(
            thrust::device,
            subs.begin(), subs.end(), 
            CosetInitializer(cosets));

    // generate initial relation table rows for coset 0
    gen_rows(0, rels, rows);

    // these keep track of what progress has been made
    int coset = 0;
    int hint = 0;

    // will break out later
    while (true) {
        // reset learning=true for all rows.
        thrust::for_each(
                thrust::device, 
                rows.begin(), 
                rows.end(),
                Relearn());

        // create a solver and apply it until nothing is being learned
        Solver solve(ngens, cosets, rels);
        while (true) {
            thrust::for_each(
                    thrust::device,
                    rows.begin(), rows.end(),
                    solve);

            // if not any row is learning, then break.
            bool r = thrust::any_of(
                    thrust::device,
                    rows.begin(), rows.end(),
                    Learning());
            if (!r) break;
        }


        // fails if hint passes the end of the table. in that case, break.
        bool done = add_coset(
                ngens,
                &coset, &hint,
                cosets);
        if (done) break;

        // generate relation table rows for new coset
        gen_rows(coset, rels, rows);

        // move completed rows to the end of the list and remove.
        auto cut = thrust::partition(
                thrust::device, 
                rows.begin(), rows.end(), 
                RowIncomplete());
        rows.erase(cut, rows.end());
    }

    return cosets;
}


int main(int argc, char* argv[]) {
    // int ngens = 4;
    // std::vector<Rel> rels = {
    //     {0, 1, 3},
    //     {1, 2, 3},
    //     {2, 3, 3},

    //     {0, 2, 2},
    //     {1, 2, 2},
    //     {1, 3, 2},
    // };
    // std::vector<int> subs = {};

    int ngens = 4;
    std::vector<Rel> rels = {
        {0, 1, 4},
        {1, 2, 3},
        {2, 3, 3},

        {0, 2, 2},
        {1, 3, 2},
        {0, 3, 2},
    };
    std::vector<int> subs = {};

    thrust::host_vector<int> cosets = solve(ngens, subs, rels);

    std::cout << cosets.size() / ngens << " cosets" << std::endl;
    for (int c = 0; c < cosets.size(); c += ngens) {
        for (int g = c; g < c + ngens; g++ ) {
            std::cout << cosets[g] << " ";
        } std::cout << std::endl;
    }

    return 0;
}

