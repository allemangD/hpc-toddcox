#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>

#include <vector>
#include <iostream>

#include "util.h"

struct Row {
    int rel;

    int l, r;

    int from, to;

    __host__ __device__
    Row() : rel(0), l(0), r(0), from(0), to(0) {}

    __device__
    Row(int rel, int cos, int size) {
        l = 0;
        r = size - 1;
        
        from = to = cos;
        
        this->rel = rel;
    }
};

std::ostream &operator<<(std::ostream &o, const Row &r) {
    return o << "Row[" << r.rel << "]{" << r.l << ":" << r.from << "-" << r.to << ":" << r.r << "}";
}

struct Rel {
    int gens[2];
    int mul;
};

struct Solver {
    int ngens;
    int *cosets;
    Rel *rels;
    
    Solver(int ngens,
           thrust::device_vector<int> &cosets,
           thrust::device_vector<Rel> &rels)
        : ngens(ngens),
           cosets(thrust::raw_pointer_cast(cosets.data())),
           rels(thrust::raw_pointer_cast(rels.data())) {
        }
    
    __device__
    void operator()(Row &r) {
        if (r.l + 1 >= r.r) return;
        
        while ((r.r - r.l) > 0) {
            int gen = rels[r.rel].gens[r.l & 1];
            int next = cosets[r.from * ngens + gen];
            if (next < 0) break;
            r.l++;
            r.from = next;
        }

        while ((r.r - r.l) > 0) {
            int gen = rels[r.rel].gens[r.r & 1];
            int next = cosets[r.to * ngens + gen];
            if (next < 0) break;
            r.r--;
            r.to = next;
        }
            
        if (r.r - r.l == 0) {
            int gen = rels[r.rel].gens[r.l & 1];
            cosets[r.from * ngens + gen] = r.to;
            cosets[r.to * ngens + gen] = r.from;
        }
    }
};

struct CosetInitializer {
    int *cosets;

    CosetInitializer(thrust::device_vector<int> &cosets)
        : cosets(thrust::raw_pointer_cast(cosets.data())) {
    }

    __device__
    void operator()(int gen) {
        cosets[gen] = 0;
    }
};

struct RowGen {
    Rel *rels;

    int coset;

    RowGen(int coset, thrust::device_vector<Rel> &rels) 
        : coset(coset),
          rels(thrust::raw_pointer_cast(rels.data())) {}

    __device__
    Row operator()(int rel) {
        return Row(rel, coset, rels[rel].mul * 2);
    }
};

void add_row(
        int ngens,
        thrust::device_vector<int> &cosets) {
    cosets.resize(cosets.size() + ngens, -1);
}

// todo: this part is _real_ slow.
void add_coset(
        int ngens,
        int *coset,
        int *hint,
        thrust::device_vector<int> &cosets) {
    *coset = cosets.size() / ngens;

    add_row(ngens, cosets);
    
    // todo: this part especially.
    while (cosets[*hint] >= 0)  *hint++;
    int from = *hint / ngens;
    int gen = *hint % ngens;
    
    cosets[*hint] = *coset;
    cosets[*coset * ngens + gen] = from;
}

void gen_rows(
        int coset,
        thrust::device_vector<Rel> &rels,
        thrust::device_vector<Row> &rows) {
    rows.resize(rows.size() + rels.size());

    thrust::counting_iterator<int> counter(0);
    thrust::transform(
            thrust::device,
            counter, counter + rels.size(),
            rows.end() - rels.size(),
            RowGen(coset, rels));
}

thrust::device_vector<int> solve(
        int ngens,
        thrust::device_vector<int> subs,
        thrust::device_vector<Rel> rels) {
    
    thrust::device_vector<int> cosets;
    thrust::device_vector<Row> rows;

    add_row(ngens, cosets);
    thrust::for_each(
            thrust::device,
            subs.begin(), subs.end(), 
            CosetInitializer(cosets));

    gen_rows(0, rels, rows);

    int coset = 0;
    int hint = 0;

    // the main loop should go here.

    std::cout << thrust::host_vector<Row>(rows) << std::endl;
    
    for (int i = 0; i < 4; i++) {
        Solver solve(ngens, cosets, rels);
        thrust::for_each(rows.begin(), rows.end(), solve);
    }

    std::cout << thrust::host_vector<Row>(rows) << std::endl;

    /*

    add_coset(ngens, &coset, &hint, cosets);

    std::cout << coset << " " << hint << std::endl;

    std::cout << rows << std::endl;

    thrust::counting_iterator<int> counter(0);

    thrust::device_vector<Row> new_rows(rels.size());
    thrust::transform(counter, counter + rels.size(), new_rows.begin(),
        RowGen(lastCoset, rels));
    rows.insert(rows.begin(), new_rows.begin(), new_rows.end());

    std::cout << rows << std::endl;

    Solver solv(ngens, cosets, rels);

    std::cout << thrust::host_vector<Row>(rows) << std::endl;
    thrust::for_each(rows.begin(), rows.end(), solv);
    std::cout << thrust::host_vector<Row>(rows) << std::endl;
    */

    return cosets;
}


int main(int argc, char* argv[]) {
    int ngens = 4;
    std::vector<Rel> rels = {
        {0, 1, 4},
        {1, 2, 3},
        {2, 3, 3},

        {0, 2, 2},
        {1, 2, 2},
        {1, 3, 2},
    };
    std::vector<int> subs = {1, 3};

    thrust::host_vector<int> cosets = solve(ngens, subs, rels);

    std::cout << cosets << std::endl;

    return 0;
}

