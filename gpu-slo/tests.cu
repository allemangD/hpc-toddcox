#include "hip/hip_runtime.h"
//#include <cstdio>
//#include <cstdlib>
//
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//#include <thrust/sequence.h>
//
//#define N 50
//
//__global__
//void vector_add(float* out, float* a, float* b, int n) {
//    for(int i = 0; i < n; i++){
//        out[i] = a[i] + b[i];
//    }
//}
//
//int main(){
//    thrust::host_vector<float> a(N);
//    thrust::sequence(a.begin(), a.end());
//
//    thrust::host_vector<float> b(N);
//    thrust::sequence(b.begin(), b.end());
//    thrust::reverse(b.begin(), b.end());
//
//    for (int i = 0; i < N; ++i) {
//        printf("%.1f ", a[i]);
//    } printf("\n");
//
//    for (int i = 0; i < N; ++i) {
//        printf("%.1f ", b[i]);
//    } printf("\n");
//
//    thrust::device_vector<float> aD = a;
//    thrust::device_vector<float> bD = b;
//    thrust::device_vector<float> outD(N);
//
//    vector_add<<<1, 1>>>(
//        thrust::raw_pointer_cast(&outD[0]),
//        thrust::raw_pointer_cast(&aD[0]),
//        thrust::raw_pointer_cast(&bD[0]),
//        N);
//
//    thrust::host_vector<float> out = outD;
//
//    for (int i = 0; i < N; ++i) {
//        printf("%.1f ", out[i]);
//    } printf("\n");
//
//    return 0;
//}

#include <cstdio>
#include <cstdlib>
#include <chrono>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>

void add_proc(int *c, int *a, int *b) {
    *c = *a + *b;
}

void test_proc(){
    int a = 0;
    int b = 1;

    auto start = std::chrono::system_clock::now();

    for (int i = 0; i < 1000000; ++i) {
        add_proc(&a, &a, &b);
    }

    auto end = std::chrono::system_clock::now();

    std::chrono::duration<float, std::micro> diff = end - start;

    printf("proc: %d: 1B in %.3f micro\n", a, diff.count());
}

__global__
void add_gpu(int *c, int *a, int *b) {
    *c = *a + *b;
}

void test_gpu(){
    thrust::device_vector<int> vals(2, 0);
    vals[0] = 0;
    vals[1] = 1;
    printf(" gpu: %d: 1B in %.3f micro\n", vals[0], 0.0f);

    int *a = thrust::raw_pointer_cast(&vals[0]);
    int *b = thrust::raw_pointer_cast(&vals[1]);

    add_gpu<<<1, 1>>>(a, a, a);

    printf(" gpu: %d: 1B in %.3f micro\n", vals[0], 0.0f);
}

int main(int argc, char *argv[]) {
    test_proc();
    test_gpu();
}

