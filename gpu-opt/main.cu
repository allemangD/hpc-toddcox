#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/logical.h>

#include <vector>
#include <iostream>
#include <chrono>

#include "util.h"
#include "groups.h"

__constant__ Rel c_rels[128];
__constant__ int c_nrels[1];
__constant__ int c_ngens[1];

struct Row {
    int rel;

    int l, r;

    int from, to;

    __host__ __device__
    Row() : rel(0), l(0), r(0), from(0), to(0) {}

    __device__
    Row(int rel, int cos, int size) {
        l = 0;
        r = size - 1;
        
        from = to = cos;
        
        this->rel = rel;
    }
};

std::ostream &operator<<(std::ostream &o, const Row &r) {
    return o << "Row[" << r.rel << "]{" << r.l << ":" << r.from << "-" << r.to << ":" << r.r << "}";
}

// this performs a pass on one relation table row, applying learned data to the coset table.
struct Solver {
    int *cosets;
    
    Solver(thrust::device_vector<int> &cosets)
        : cosets(thrust::raw_pointer_cast(cosets.data())) {
    }
    
    __device__
    void operator()(Row &drow) {
        Row row = drow;

        if (row.r - row.l <= 0) {
            return;
        }
        
        while (row.r - row.l > 0) {
            int gen = c_rels[row.rel].gens[row.l & 1];
            int next = cosets[row.from * c_ngens[0] + gen];
            if (next < 0) break;
            row.l++;
            row.from = next;
        }

        while (row.r - row.l > 0) {
            int gen = c_rels[row.rel].gens[row.r & 1];
            int next = cosets[row.to * c_ngens[0] + gen];
            if (next < 0) break;
            row.r--;
            row.to = next;
        }

        drow = row;
            
        if (row.r - row.l <= 0) { 
            int gen = c_rels[row.rel].gens[row.l & 1];
            cosets[row.from * c_ngens[0] + gen] = row.to;
            cosets[row.to * c_ngens[0] + gen] = row.from;
            return;
        }
    }
};

// this sets the inital row in the coset table based on the subgroup generators
struct CosetInitializer {
    int *cosets;

    CosetInitializer(thrust::device_vector<int> &cosets)
        : cosets(thrust::raw_pointer_cast(cosets.data())) {
    }

    __device__
    void operator()(int gen) {
        cosets[gen] = 0;
    }
};

// this creates rows for cosets by index of each relation table
struct RowGen {
    int coset;

    RowGen(int coset) 
        : coset(coset) {
    }

    __device__
    Row operator()(int rel) {
        return Row(rel, coset, c_rels[rel].mul * 2);
    }
};

// determines if rows are incomplete; used to remove completed rows
struct RowIncomplete {
    __device__
    bool operator()(Row r) {
        return r.r - r.l > 1;
    }
};

// add a row to the coset table filled with -1
void add_row(
        int ngens,
        thrust::device_vector<int> &cosets) {
    cosets.resize(cosets.size() + ngens, -1);
};

// add a new coset to the coset table, picking up where the last call left off.
// todo: this part is _real_ slow.
bool add_coset(
        int ngens,
        int *coset,
        int *hint,
        thrust::device_vector<int> &dcosets) {
    int offset = *hint;
    thrust::host_vector<int> cosets(dcosets.begin() + offset, dcosets.end());
    *coset = dcosets.size() / ngens;

    // todo: this part especially.
    while (cosets[*hint - offset] >= 0) {
        *hint = *hint + 1;
        if (*hint - offset >= cosets.size()) 
            return true;
    }
    int from = *hint / ngens;
    int gen = *hint % ngens;
    
    add_row(ngens, dcosets);
    
    dcosets[*hint] = *coset;
    dcosets[*coset * ngens + gen] = from;

    return false;
}

// add a row for each relation table for some coset
void gen_rows(
        int coset,
        int nrels,
        thrust::device_vector<Row> &rows) {
    rows.resize(rows.size() + nrels);

    thrust::counting_iterator<int> counter(0);
    thrust::transform(
            thrust::device,
            counter, counter + nrels,
            rows.end() - nrels,
            RowGen(coset));
}

// do everything. data is implicitly passed to the device via device_vector.
thrust::device_vector<int> solve(
        int ngens,
        int nrels,
        thrust::device_vector<int> subs) {
    
    thrust::device_vector<int> cosets;
    thrust::device_vector<Row> rows;

    // create the inital row and populate it from subs
    add_row(ngens, cosets);
    thrust::for_each(
            thrust::device,
            subs.begin(), subs.end(), 
            CosetInitializer(cosets));

    // generate initial relation table rows for coset 0
    gen_rows(0, nrels, rows);

    // these keep track of what progress has been made
    int coset = 0;
    int hint = 0;

    // will break out later
    while (true) {
        // create a solver and apply it until nothing is being learned
        Solver solve(cosets);
        thrust::for_each(
                thrust::device,
                rows.begin(), rows.end(),
                solve);

        // fails if hint passes the end of the table. in that case, break.
        bool done = add_coset(
                ngens,
                &coset, &hint,
                cosets);
        if (done) break;

        // generate relation table rows for new coset
        gen_rows(coset, nrels, rows);

        // move completed rows to the end of the list and remove.
        auto cut = thrust::partition(
                thrust::device, 
                rows.begin(), rows.end(), 
                RowIncomplete());
        rows.erase(cut, rows.end());
    }

    return cosets;
}


int main(int argc, const char* argv[]) {
    Coxeter cox;
    cox = proc_args(argc, argv);
    std::vector<int> subs = {};
    int nrels = cox.rels.size();
    int ngens = cox.ngens;

    hipMemcpyToSymbol(HIP_SYMBOL(c_ngens), &ngens, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_nrels), &nrels, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_rels), cox.rels.data(), cox.rels.size() * sizeof(Rel));

    auto s = std::chrono::system_clock::now();
    thrust::host_vector<int> cosets = solve(cox.ngens, nrels, subs);
    auto e = std::chrono::system_clock::now();

    std::chrono::duration<float> diff = e - s;
    int order = cosets.size() / cox.ngens;

    // type, arg, ngens, time, order
    std::cout << cox.ngens << ',' << diff.count() << ',' << order << std::endl;

    return 0;
}

